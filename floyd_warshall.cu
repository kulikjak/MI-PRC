#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// #define _CHECK_MATRICES
// #define _PRINT_RESULT
#define _KERNEL 0

#define TILE_SIZE 32
#define LOG_TILE_SIZE 5

clock_t _start_in, _start_out;
clock_t _end_in, _end_out;

// Kernel for independent blocks
__global__ void kernel_independent_blocks(matrix __dm, int32_t __size,
                                          int32_t __b, int32_t k) {
  // get i and j from block and thread information
  const int32_t i = blockIdx.x + (__b << LOG_TILE_SIZE);
  const int32_t j = threadIdx.x + (__b << LOG_TILE_SIZE);

  // check boundaries for matrices with indivisible size
  if (i >= __size || j >= __size) return;

  // calculate values
  __dm[i][j] = (__dm[i][k] + __dm[k][j] < __dm[i][j]) ? __dm[i][k] + __dm[k][j]
                                                      : __dm[i][j];
}

// Kernel for i-aligned singly depenent blocks
__global__ void kernel_line_dependent_blocks(matrix __dm, int32_t __size,
                                             int32_t __b, int32_t k) {
  // get i, j and ib from block and thread information
  const int32_t ib = blockIdx.x;

  const int32_t i = threadIdx.x + (__b << LOG_TILE_SIZE);
  const int32_t j = threadIdx.y + (ib << LOG_TILE_SIZE);

  // skip already calculated tile
  if (__b == ib) return;

  // check boundaries for matrices with indivisible size
  if (i >= __size || j >= __size) return;

  // calculate values
  __dm[i][j] = (__dm[i][k] + __dm[k][j] < __dm[i][j]) ? __dm[i][k] + __dm[k][j]
                                                      : __dm[i][j];
}

// Kernel for j-aligned singly depenent blocks
__global__ void kernel_column_dependent_blocks(matrix __dm, int32_t __size,
                                               int32_t __b, int32_t k) {
  // get i, j and jb from block and thread information
  const int32_t jb = blockIdx.x;

  const int32_t i = threadIdx.x + (jb << LOG_TILE_SIZE);
  const int32_t j = threadIdx.y + (__b << LOG_TILE_SIZE);

  // skip already calculated tile
  if (jb == __b) return;

  // check boundaries for matrices with indivisible size
  if (i >= __size || j >= __size) return;

  // calculate values
  __dm[i][j] = (__dm[i][k] + __dm[k][j] < __dm[i][j]) ? __dm[i][k] + __dm[k][j]
                                                      : __dm[i][j];
}

// Kernel for double depenent blocks (very simple)
__global__ void kernel_double_dependent_blocks(matrix __dm, int32_t __size,
                                               int32_t __b, int32_t k) {
  // get i, j, ib and jb from block and thread information
  const int32_t ib = blockIdx.x;
  const int32_t jb = blockIdx.y;

  const int32_t i = threadIdx.x + (jb << LOG_TILE_SIZE);
  const int32_t j = threadIdx.y + (ib << LOG_TILE_SIZE);

  // skip already calculated tile
  if (ib == __b || jb == __b) return;

  // check boundaries for matrices with indivisible size
  if (i >= __size || j >= __size) return;

  // calculate values
  __dm[i][j] = (__dm[i][k] + __dm[k][j] < __dm[i][j]) ? __dm[i][k] + __dm[k][j]
                                                      : __dm[i][j];
}

// Kernel for double depenent blocks (threads on whole line)
__global__ void kernel_l_double_dependent_blocks(matrix __dm, int32_t __size,
                                                 int32_t __b) {
  int32_t k, j;

  // Get some basic variables
  const int32_t ib = blockIdx.x;
  const int32_t jb = blockIdx.y;

  // Real position in the grid
  const int32_t ri = (ib << LOG_TILE_SIZE) + threadIdx.x;
  const int32_t rj = (jb << LOG_TILE_SIZE);

  // skip already calculated tile
  if (ib == __b || jb == __b) return;

  // check boundaries for matrices with indivisible size
  if (ri >= __size) return;

  for (k = __b * TILE_SIZE; k < (__b + 1) * TILE_SIZE; k++) {
    if (k >= __size) break;
    for (j = 0; j < TILE_SIZE; j++) {
      if (rj + j >= __size) break;
      // calculate values
      __dm[ri][rj + j] = (__dm[ri][k] + __dm[k][rj + j] < __dm[ri][rj + j])
                             ? __dm[ri][k] + __dm[k][rj + j]
                             : __dm[ri][rj + j];
    }
  }
}


// Kernel for double depenent blocks (shared current block)
__global__ void kernel_sm_double_dependent_blocks(matrix __dm, int32_t __size,
                                                  int32_t __b) {
  int32_t k;

  // Get some basic variables
  const int32_t ib = blockIdx.x;
  const int32_t jb = blockIdx.y;

  // Real position in the grid
  const int32_t ri = threadIdx.x + (jb << LOG_TILE_SIZE);
  const int32_t rj = threadIdx.y + (ib << LOG_TILE_SIZE);

  // Position in shared memory grid
  const int32_t i = threadIdx.x;
  const int32_t j = threadIdx.y;

  // skip already calculated tile
  if (ib == __b || jb == __b) return;

  // check boundaries for matrices with indivisible size
  if (ri >= __size || rj >= __size) return;

  __shared__ int32_t block[TILE_SIZE][TILE_SIZE];

  // Copy data into shared memory
  block[i][j] = __dm[ri][rj];
  __syncthreads();

  for (k = __b * TILE_SIZE; k < (__b + 1) * TILE_SIZE; k++) {
    if (k >= __size) break;
    // calculate values
    block[i][j] = (__dm[ri][k] + __dm[k][rj] < block[i][j])
                      ? __dm[ri][k] + __dm[k][rj]
                      : block[i][j];
  }

  // Copy data back from shared into global memory
  __dm[ri][rj] = block[i][j];
  __syncthreads();
}

// Kernel for double depenent blocks (threads on whole line with shared memory block)
__global__ void kernel_sml_double_dependent_blocks(matrix __dm, int32_t __size,
                                                   int32_t __b) {
  int32_t k, j;

  // Get some basic variables
  const int32_t ib = blockIdx.x;
  const int32_t jb = blockIdx.y;

  // Real position in the grid
  const int32_t ri = (ib << LOG_TILE_SIZE) + threadIdx.x;
  const int32_t rj = (jb << LOG_TILE_SIZE);

  // Position in shared memory grid
  const int32_t i = threadIdx.x;

  // skip already calculated tile
  if (ib == __b || jb == __b) return;

  // check boundaries for matrices with indivisible size
  if (ri >= __size) return;

  __shared__ int32_t block[TILE_SIZE][TILE_SIZE];

  // Copy data into shared memory
  for (j = 0; j < TILE_SIZE; j++) {
    if (rj + j >= __size) break;
    block[i][j] = __dm[ri][rj + j];
  }
  __syncthreads();

  for (k = __b * TILE_SIZE; k < (__b + 1) * TILE_SIZE; k++) {
    __syncthreads();
    if (k >= __size) continue;
    for (j = 0; j < TILE_SIZE; j++) {
      if (rj + j >= __size) break;

      block[i][j] = (__dm[ri][k] + __dm[k][rj + j] < block[i][j])
                        ? __dm[ri][k] + __dm[k][rj + j]
                        : block[i][j];
    }
  }

  // Copy data back from shared into global memory
  for (j = 0; j < TILE_SIZE; j++) {
    if (rj + j >= __size) break;
    __dm[ri][rj + j] = block[i][j];
  }
  __syncthreads();
}

void run_algorithm(matrix __dm, int32_t __size) {
  const int32_t s = TILE_SIZE;
  const int32_t tile_count = (__size + s - 1) / s;

  const dim3 tile2D(TILE_SIZE, TILE_SIZE);
  const dim3 grid2D(tile_count, tile_count);

  int32_t k, b;

  for (b = 0; b < tile_count; b++) {
    // Process the independent block first
    for (k = b * s; k < (b + 1) * s; k++) {
      if (k >= __size) break;
      kernel_independent_blocks<<<s, s>>>(__dm, __size, b, k);
      HANDLE_ERROR(hipDeviceSynchronize());
    }

    // i-aligned singly depenent blocks
    for (k = b * s; k < (b + 1) * s; k++) {
      if (k >= __size) break;
      kernel_line_dependent_blocks<<<tile_count, tile2D>>>(__dm, __size, b, k);
      HANDLE_ERROR(hipDeviceSynchronize());
    }

    // j-aligned singly depenent blocks
    for (k = b * s; k < (b + 1) * s; k++) {
      if (k >= __size) break;
      kernel_column_dependent_blocks<<<tile_count, tile2D>>>(__dm, __size, b, k);
      HANDLE_ERROR(hipDeviceSynchronize());
    }

    // double dependent blocks
    switch (_KERNEL) {
      case 0:
        for (k = b * s; k < (b + 1) * s; k++) {
          if (k >= __size) break;
          kernel_double_dependent_blocks<<<grid2D, tile2D>>>(__dm, __size, b, k);
          HANDLE_ERROR(hipDeviceSynchronize());
        }
        break;

      case 1:
        kernel_l_double_dependent_blocks<<<grid2D, tile2D>>>(__dm, __size, b);
        HANDLE_ERROR(hipDeviceSynchronize());
        break;

      case 2:
        kernel_sm_double_dependent_blocks<<<grid2D, tile2D>>>(__dm, __size, b);
        HANDLE_ERROR(hipDeviceSynchronize());
        break;

      case 3:
        kernel_sml_double_dependent_blocks<<<grid2D, tile2D>>>(__dm, __size, b);
        HANDLE_ERROR(hipDeviceSynchronize());
        break;

      default:
        printf("Wrong Kernel specified.\n");
        exit(EXIT_FAILURE);
    }
  }
}

int main(int argc, char* argv[]) {
  int32_t size;

  FILE* graph_file;
  matrix graph_mtx;
  matrix hostMtx, devMtx;

  if (argc != 2) {
    printf("Wrong input\n");
    printf("Usage: %s GRAPH_FILE\n", argv[0]);
    return EXIT_FAILURE;
  }

  graph_file = fopen(argv[1], "r");
  if (graph_file == NULL) {
    printf("Cannot open input file.\n");
    return EXIT_FAILURE;
  }

  if (fscanf(graph_file, "%d", &size) == EOF) {
    printf("Input file is empty.\n");
    return EXIT_FAILURE;
  }

  graph_mtx = read_matrix(graph_file, size);

  hostMtx = get_distance_matrix(graph_mtx, size);

  _start_out = clock();
  devMtx = allocate_and_init_matrix_GPU(hostMtx, size);
  _start_in = clock();

  run_algorithm(devMtx, size);

  _end_in = clock();
  copy_GPU_to_CPU(hostMtx, devMtx, size);
  _end_out = clock();

#ifdef _CHECK_MATRICES

  printf("Checking that parallel algorithm runs correctly\n");

  matrix dist_mtx2 = get_distance_matrix(graph_mtx, size);
  dist_mtx2 = floyd_warshall_seq(dist_mtx2, size);
  if (compare_matrices(hostMtx, dist_mtx2, size)) {
    printf("Both parallel and serial result matrices match.\n");
  } else {
    printf("Serial and parallel result matrices are different.\n");
    printf("Something is wrong!!!.\n");
  }
  free_matrix_CPU(dist_mtx2, size);

#endif

#ifdef _PRINT_RESULT

  print_matrix(hostMtx, size);

#endif

  free_matrix_GPU(devMtx, size);

  free_matrix_CPU(hostMtx, size);
  free_matrix_CPU(graph_mtx, size);
  fclose(graph_file);

  printf("== Time: %lf (without data copy)\n",
         double(_end_in - _start_in) / CLOCKS_PER_SEC);
  printf("== Time: %lf (with data copy)\n",
         double(_end_out - _start_out) / CLOCKS_PER_SEC);

  return EXIT_SUCCESS;
}
